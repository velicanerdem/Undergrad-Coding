#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "common.h"
#include <omp.h>


#define TRAIN_NUM 19000
#define TEST_NUM 1000
#define DIMENSIONS 16

#define THREADS_PER_BLOCK 256
#define N TRAIN_NUM + THREADS_PER_BLOCK - 1

__global__ void vector_distance(int *a, int *b, int *c) {
  int index = (THREADS_PER_BLOCK * blockIdx.x) + threadIdx.x;
  if (index < TRAIN_NUM){
	  int dim;
	  int distance = 0;
	  int one_dim_dif;
	  for (dim = 0; dim < 16; ++dim){
		 one_dim_dif = a[index * DIMENSIONS + dim] - b[dim];
		 distance += one_dim_dif * one_dim_dif;
	  }
	  //stores index as the point in train
	  c[index] = distance;
  }
}

int main(){
	
	FILE * train_file;
	FILE * test_file;
	FILE * output_file;
	train_file = fopen("train.txt", "r");
	test_file = fopen("test.txt", "r");
	
	int train_data[TRAIN_NUM * DIMENSIONS];
	int test_data[TEST_NUM][DIMENSIONS];
	int output[TEST_NUM];
	
	int i,j;
	
	char singleLine[150];
	char * cdim;
	char * str;
	int dim;
	int line = 0;
	
	while (!feof(train_file)){
		dim = 0;
		fgets(singleLine, 150, train_file);
		str = strdup(singleLine);
		while (cdim = strsep(&str, ",")){
			train_data[line * DIMENSIONS + dim++] = atoi(cdim);
		}
		++line;
	}
	
	fclose(train_file);
	
	line = 0;
	while (!feof(test_file)){
		dim = 0;
		fgets(singleLine, 150, test_file);
		str = strdup(singleLine);
		while (cdim = strsep(&str, ",")){
			test_data[line][dim++] = atoi(cdim);
		}
		++line;
	}
	
	fclose(test_file);
	
	size_t size_train = TRAIN_NUM * DIMENSIONS * sizeof(int);
	size_t size_dist  = TRAIN_NUM * sizeof(int);
	size_t size_testpoint  = DIMENSIONS * sizeof(int);
	
	int * dist;
	hipHostMalloc( (void**) &dist, size_dist);

	int * d_train, * d_dist, * d_testpoint;
	
	double run_time;
	double start_time = omp_get_wtime();
	
	cudaCheck(hipMalloc( (void **) &d_train, size_train));
	cudaCheck(hipMalloc( (void **) &d_dist, size_dist));
	cudaCheck(hipMalloc( (void **) &d_testpoint, size_testpoint));
	
	cudaCheck(hipPeekAtLastError());
	
	int min, point_min;
	cudaCheck(hipMemcpy(d_train, train_data, size_train, hipMemcpyHostToDevice));
	for (i = 0; i < TEST_NUM; ++i){
		point_min = -1;
		min = 999999;
		cudaCheck(hipMemcpy(d_testpoint, test_data[i], size_testpoint, hipMemcpyHostToDevice));
		vector_distance<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>
			(d_train, d_testpoint, d_dist);
		cudaCheck(hipMemcpy(dist, d_dist, size_dist, hipMemcpyDeviceToHost));
		for (j = 0; j < TRAIN_NUM; ++j){
			if (dist[j] < min){
				min = dist[j];
				point_min = j;
			}
		}
		output[i] = point_min;
	}
	
	run_time = omp_get_wtime() - start_time;
	printf("Total time: %6.3f\n", run_time);
	
	output_file = fopen("output.txt", "w");
	
	for (i = 0; i < TRAIN_NUM; ++i)
		fprintf(output_file, "%d\n", output[i]);

	
	fclose(output_file);
	
	hipHostFree(dist);
	
	hipFree(d_train);
	hipFree(d_testpoint);
	hipFree(d_dist);
}
